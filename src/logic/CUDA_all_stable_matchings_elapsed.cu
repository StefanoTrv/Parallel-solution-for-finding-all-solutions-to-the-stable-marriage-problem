#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <chrono>
#include <stdint.h>
#include "..\utilities\utilities.h"
#include "..\utilities\handle_error.cuh"
#include "..\data_structures\data_structures.h"


#define min(i, j) (((i) < (j)) ? (i) : (j))
#define max(i, j) (((i) > (j)) ? (i) : (j))


struct ResultsList* all_stable_matchings_times_CUDA(int n, int* men_preferences, int* women_preferences, int* time_gale_shapley, int* time_find_all_rotations, int* time_overhead, int* time_kernel, int* time_overhead_kernel, int* time_recursive, int* time_total){
	// Time measure
    std::chrono::steady_clock::time_point start_time;
    std::chrono::steady_clock::time_point end_time;
	std::chrono::steady_clock::time_point start_time_kernel;
    std::chrono::steady_clock::time_point end_time_kernel;

	struct ResultsList* results_list = (struct ResultsList*) malloc(sizeof (struct ResultsList));
	
	start_time = std::chrono::steady_clock::now();
	int* top_matching = gale_shapley(n,men_preferences,women_preferences);
	int* inverted_bottom_matching = gale_shapley(n, women_preferences, men_preferences);
	end_time = std::chrono::steady_clock::now();
	*time_gale_shapley = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();
	*time_total += *time_gale_shapley;

	int* bottom_matching = (int*)malloc(sizeof (int) * n);
	for(int i = 0; i < n; i++){
		bottom_matching[inverted_bottom_matching[i]] = i;
	}
	free(inverted_bottom_matching);

	//termina subito se non ci sono rotazioni
	int only_one_matching=1;
	for(int i=0;i<n;i++){
		if(top_matching[i]!=bottom_matching[i]){
			only_one_matching=0;
			break;
		}
	}
	if(only_one_matching){
		results_list->first = (struct ResultsListElement*) malloc(sizeof (struct ResultsListElement));
		results_list->first->value = top_matching;
		results_list->first->next = NULL;
		results_list->last = results_list->first;
		free(bottom_matching);
		return results_list;
	}
	
	//copia top_matching
	int* top_matching_copy = (int*) malloc(sizeof (int) * n);
	for(int i = 0; i < n; i++){
		top_matching_copy[i] = top_matching[i];
	}

	//crea la lista delle rotazioni
	start_time = std::chrono::steady_clock::now();
	struct RotationsList* rotations_list = find_all_rotations(men_preferences, women_preferences, n, top_matching_copy,bottom_matching);
	end_time = std::chrono::steady_clock::now();
	*time_find_all_rotations = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();
	*time_total += *time_find_all_rotations;
	free(bottom_matching);

	//crea il grafo delle rotazioni
	start_time = std::chrono::steady_clock::now();

	//SEZIONE PARALLELIZZATA
	//creazione delle strutture dati di input
	start_time = std::chrono::steady_clock::now();
	int number_of_rotations = 0;
	int total_number_of_pairs = 0;
	struct RotationsListElement* list_el = rotations_list->first;
	struct RotationList* rotation_el;
	while(list_el!=NULL){
		number_of_rotations++;
		rotation_el=list_el->value->rotation;
		while(rotation_el!=NULL){
			total_number_of_pairs++;
			rotation_el=rotation_el->next;
		}
		list_el=list_el->next;
	}

	int* rotations_vector;
	HANDLE_ERROR(hipHostAlloc((void**)&rotations_vector, sizeof (int) * total_number_of_pairs * 2, hipHostMallocMapped));
	int* end_displacement_vector;
	HANDLE_ERROR(hipHostAlloc((void**)&end_displacement_vector, sizeof (int) * number_of_rotations, hipHostMallocMapped));
	struct RotationNode** rotation_vector = (struct RotationNode**)malloc(sizeof (struct RotationNode*) * number_of_rotations); //per velocizzare il salvataggio dei risultati

	list_el = rotations_list->first;
	int c1 = 0;
	int c2 = 0;
	while(list_el!=NULL){
		rotation_vector[list_el->value->index]=list_el->value;//riempio rotation_vector
		rotation_el=list_el->value->rotation;
		while(rotation_el!=NULL){//salva tutte le coppie
			rotations_vector[c1]=rotation_el->man;
			rotations_vector[total_number_of_pairs+c1]=rotation_el->woman;
			c1++;
			rotation_el=rotation_el->next;
		}
		c2=c1-1;
		end_displacement_vector[list_el->value->index]=c2; //il displacement di questa rotazione
		list_el=list_el->next;
	}

	//preparazione per il lancio del kernel
	int* triangular_matrix, *dev_triangular_matrix, *dev_rotations_vector, *dev_end_displacement_vector, *dev_top_matching, *dev_men_preferences, *dev_women_preferences;

	HANDLE_ERROR(hipHostAlloc((void**)&triangular_matrix, sizeof (int) * ((number_of_rotations-1)*number_of_rotations)/2, hipHostMallocMapped));
	
	if(number_of_rotations>1){
		(hipHostGetDevicePointer(&dev_triangular_matrix, triangular_matrix, 0));
	} else{
		dev_triangular_matrix=NULL;
	}
	HANDLE_ERROR(hipHostGetDevicePointer(&dev_rotations_vector, rotations_vector, 0));
	HANDLE_ERROR(hipHostGetDevicePointer(&dev_end_displacement_vector, end_displacement_vector, 0));

	HANDLE_ERROR(hipMalloc((void**)&dev_top_matching, sizeof(int) * n));
	HANDLE_ERROR(hipMalloc((void**)&dev_men_preferences, sizeof(int) * n * n));
	HANDLE_ERROR(hipMalloc((void**)&dev_women_preferences, sizeof(int) * n * n));

	HANDLE_ERROR(hipMemcpy(dev_top_matching, top_matching, sizeof(int) * n, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_men_preferences, men_preferences, sizeof(int) * n * n, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_women_preferences, women_preferences, sizeof(int) * n * n, hipMemcpyHostToDevice));

	//lancio del kernel
	int NumThPerBlock = min(max(number_of_rotations, n), 1024);
	size_t free_t,total_t;
	hipMemGetInfo(&free_t,&total_t);
	hipDeviceSetLimit(hipLimitMallocHeapSize, free_t * 0.9);
	start_time_kernel = std::chrono::steady_clock::now();
	build_graph_CUDA<<<1, NumThPerBlock>>>(n, number_of_rotations, total_number_of_pairs, dev_rotations_vector, dev_end_displacement_vector,  dev_top_matching, dev_women_preferences, dev_men_preferences, dev_triangular_matrix);

	//libero memoria
	hipDeviceSynchronize();
	end_time_kernel = std::chrono::steady_clock::now();
	*time_kernel = std::chrono::duration_cast<std::chrono::milliseconds>(end_time_kernel - start_time_kernel).count();

	HANDLE_ERROR(hipFree(dev_top_matching));
	HANDLE_ERROR(hipFree(dev_men_preferences));
	HANDLE_ERROR(hipFree(dev_women_preferences));

	//applico i risultati alle strutture dati dell'host
	list_el = rotations_list->first;
	int y;
	struct SuccessorsList* sl_el;
	while(list_el!=NULL){
		y=list_el->value->index;
		for(int x = 0; x<y; x++){
			if(triangular_matrix[(y-1)*number_of_rotations+x]){//se y dipende da x
				list_el->value->missing_predecessors++;//incremento il numero di predecessori di y
				//e aggiungo y tra i successori di x
				sl_el=(struct SuccessorsList*)malloc(sizeof (struct SuccessorsList));
				sl_el->value=list_el->value;
				sl_el->next=rotation_vector[x]->successors;
				rotation_vector[x]->successors=sl_el;
			}
		}
		list_el=list_el->next;
	}

	//libero memoria
	HANDLE_ERROR(hipHostFree(triangular_matrix));
	HANDLE_ERROR(hipHostFree(rotations_vector));
	HANDLE_ERROR(hipHostFree(end_displacement_vector));
	free(rotation_vector);

	end_time = std::chrono::steady_clock::now();
	*time_overhead_kernel = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();
	*time_overhead = *time_overhead_kernel - *time_kernel;
	*time_total += *time_overhead_kernel;
	//FINE SEZIONE PARALLELIZZATA

	//calcolo la lista delle rotazioni libere
	struct RotationsList* free_rotations_list = (struct RotationsList*)malloc(sizeof (struct RotationsList));
	free_rotations_list->first=NULL;
	free_rotations_list->last=NULL;
	list_el = rotations_list->first;
	while(list_el!=NULL){
		if(list_el->value->missing_predecessors==0){
			appendRotationsList(free_rotations_list,list_el->value);
		}
		list_el=list_el->next;
	}
	
	//aggiungo top matching ai risultati
	results_list->first = (struct ResultsListElement*) malloc(sizeof (struct ResultsListElement));
	for(int i = 0; i < n; i++){ //per non lavorare sul matching salvato tra i risultati
		top_matching_copy[i] = top_matching[i];
	}
	results_list->first->value = top_matching_copy;
	results_list->first->next = NULL;
	results_list->last = results_list->first;

	if(rotations_list->first != NULL){
		start_time = std::chrono::steady_clock::now();
		recursive_search(top_matching, n, free_rotations_list->first, results_list);
		end_time = std::chrono::steady_clock::now();
		*time_recursive = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();
		*time_total += *time_recursive;
	}
	
	free(top_matching);
	free_rotations_list_struct(rotations_list);
	list_el=free_rotations_list->first;
	struct RotationsListElement* temp;
	while(list_el!=NULL){
		temp=list_el;
		list_el=list_el->next;
		free(temp);
	}
	free(free_rotations_list);
	return results_list;
}