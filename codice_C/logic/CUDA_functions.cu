#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "..\data_structures\data_structures.h"

#define false 0
#define true 1


__global__ void build_graph_CUDA(int, int, int, int*, int*, int*, int*, int*, int*);

__device__ int* label_matrix;
__device__ int* is_stable_matrix;
__device__ int* label_second_condition;
__device__ int* applied_rotations;
__shared__ int* first_men_preferences_index;
__shared__ int* first_women_preferences_index;

__global__ void build_graph_CUDA(int n, int number_of_rotations, int total_number_of_pairs, int* rotations_vector, int* end_displacement_vector, int* top_matching, int* women_preferences, int* men_preferences, int* triangular_matrix){
	int woman, i, j, first_woman, man, next_woman, k, p_star, iterations;
	if(threadIdx.x==0){
		label_matrix = (int*)malloc(sizeof (int) * n * n);
		is_stable_matrix = (int*)malloc(sizeof (int) * n * n);
		label_second_condition = (int*)malloc(sizeof (int) * n * n);
		applied_rotations = (int*)malloc(sizeof (int) * number_of_rotations * n);
		first_men_preferences_index = (int*)malloc(sizeof (int) * n);
		first_women_preferences_index = (int*)malloc(sizeof (int) * n);
	}
	__syncthreads();

	for(i=threadIdx.x;i<n;i+=blockDim.x){
		for(j=0; j<n; j++){
			label_matrix[i*n+j]=n;
			is_stable_matrix[j*n+i]=false;
			label_second_condition[j*n+i]=false;
		}
		woman = top_matching[i];
	}
	__syncthreads();

	for(i=threadIdx.x;i<n;i+=blockDim.x){
		is_stable_matrix[top_matching[i]*n+i]=true;
		for(j=0;j<n;j++){
			if(top_matching[j]==men_preferences[j*n+i]) first_men_preferences_index[j]=i;
			if(j==women_preferences[top_matching[j]*n+i]) first_women_preferences_index[top_matching[j]]=i;
		}
	}
	__syncwarp();

	for (i=threadIdx.x;i<number_of_rotations;i+=blockDim.x){
		for(j=0;j<n;j++){
			applied_rotations[j*n+i]=false;
		}
	}
	__syncwarp();

	for (i = threadIdx.x; i < ((number_of_rotations-1)*number_of_rotations)/2; i+=blockDim.x) {
		triangular_matrix[i] = false;
		//printf("%i ",triangular_matrix[i]);
	}
	__syncwarp();

	iterations = number_of_rotations/blockDim.x + ((number_of_rotations%blockDim.x < 1) ? 0 : 1);
	for (i=threadIdx.x;i<iterations*blockDim.x;i+=blockDim.x){
		if(i<number_of_rotations){
			if(i==0){
				j=0;
			}else{
				j=end_displacement_vector[i-1]+1;
			}
			printf("\nThread: %i\ti: %i\tj: %i",threadIdx.x,i,j);
		} else {
			printf("\nThread: %i\ti: %i\tNON ESEGUITO",threadIdx.x,i);
		}
		__syncwarp();
		if(i<number_of_rotations){
			first_woman=rotations_vector[total_number_of_pairs+j];
			for(;j<=end_displacement_vector[i];j++){
				man=rotations_vector[j];
				if(j==end_displacement_vector[i]){
					next_woman=first_woman;
				}else{
					next_woman=rotations_vector[total_number_of_pairs+j+1];
				}
				woman=rotations_vector[total_number_of_pairs+j];
				printf("\nCoppia della rotazione %i: (%i,%i).\t(i=%i, j=%i, w_index=%i)",i,man,woman,i,j,total_number_of_pairs+j);
				//aggiorna rispetto alla donna
				k=first_women_preferences_index[next_woman]-1;
				printf("\nkw[%i] = %i",next_woman,k);
				while(women_preferences[next_woman*n+k]!=man){
					atomicMin(label_matrix + (next_woman*n+women_preferences[next_woman*n+k]),i);// => label_matrix[next_woman*n+women_preferences[next_woman*n+k]]=i;
					printf("\nlabel_matrix[%i = (%i, %i)] = %i\t(type 2)",next_woman*n+women_preferences[next_woman*n+k],next_woman,women_preferences[next_woman*n+k],label_matrix[next_woman*n+women_preferences[next_woman*n+k]]);
					k--;
				}
				//aggiorna rispetto all'uomo
				k=first_men_preferences_index[man]+1;
				printf("\nkm[%i] = %i",man,k);
				while(men_preferences[man*n+k]!=next_woman){
					label_second_condition[man*n+men_preferences[man*n+k]]=true;
					printf("\n\tlabel_second_condition[%i = (%i,%i)] = %i\t(type 2)",man*n+men_preferences[man*n+k],man,men_preferences[man*n+k],label_second_condition[man*n+men_preferences[man*n+k]]);
					k++;
				}

				is_stable_matrix[next_woman*n+man]=true;
				printf("\nLa coppia uomo %i e donna %i e' stabile.",man, woman);
				atomicMin(label_matrix + (woman*n+man),i);// => label_matrix[woman*n+man]=i;
				printf("\nlabel_matrix[%i = (%i,%i)] = %i\t(type 1)",(woman*n+man),woman,man,label_matrix[(woman*n+man)]);
			}
		}
		__syncwarp();
	}
	__syncthreads();
	if(threadIdx.x==0){
		printf("\nLABEL_MATRIX:\n");
		for(i=0;i<n;i++){
			for(j=0; j<n; j++){
				printf("%i\t",label_matrix[i*n+j]);
			}
			printf("\n");
		}
	}
	if(threadIdx.x==0){
		printf("\nIS STABLE MATRIX:\n");
		for(i=0;i<n;i++){
			for(j=0; j<n; j++){
				printf("%i\t",is_stable_matrix[i*n+j]);
			}
			printf("\n");
		}
	}
	if(threadIdx.x==0){
		printf("\nLABEL SECOND CONDITION:\n");
		for(i=0;i<n;i++){
			for(j=0; j<n; j++){
				printf("%i\t",label_second_condition[i*n+j]);
			}
			printf("\n");
		}
	}

	iterations = n/blockDim.x + ((n%blockDim.x < 1) ? 0 : 1);
	for(man=threadIdx.x;man<iterations*blockDim.x;man+=blockDim.x){
		if(man<n){
			k=0;
			while(top_matching[man]!=men_preferences[man*n+k]){
				k++;
			}
			//printf("\n2. Thread: %i\tman: %i\tk: %i\tm_p[]: %i",threadIdx.x,man,k,men_preferences[man*n+k]);
		} else {
			//printf("\n2. Thread: %i\tman: %i\tNON ESEGUITO",threadIdx.x,man);
		}
		__syncwarp();
		if(man<n){
			p_star=-1;
			for(j=k;j<n;j++){
				printf("Thread %i, man = %i, j = %i, p_star = %i, label_matrix[%i] = %i\n",threadIdx.x,man,j,p_star,woman*n+man,label_matrix[woman*n+man]);
				woman = men_preferences[man*n+j];
				if(label_matrix[woman*n+man]==n) continue;
				if(is_stable_matrix[woman*n+man]){//label di tipo 1
					if(p_star!=-1){
						triangular_matrix[(label_matrix[woman*n+man]-1)*number_of_rotations+p_star]=true;
						printf("triangular_matrix[%i] = %i (type 1, thread: %i)\tda %i a %i\n",(label_matrix[woman*n+man]-1)*number_of_rotations+p_star,triangular_matrix[(label_matrix[woman*n+man]-1)*number_of_rotations+p_star],threadIdx.x,p_star,label_matrix[woman*n+man]);
					}
					p_star=label_matrix[woman*n+man];
					applied_rotations[man*n+label_matrix[woman*n+man]]=true;
				} else if(!applied_rotations[man*n+label_matrix[woman*n+man]] && label_second_condition[man*n+woman]){//label di tipo 2
					triangular_matrix[(p_star-1)*number_of_rotations+label_matrix[woman*n+man]]=true;
					printf("triangular_matrix[%i] = %i  (type 2, thread: %i)\tda %i a %i\n",(p_star-1)*number_of_rotations+label_matrix[woman*n+man],triangular_matrix[(p_star-1)*number_of_rotations+label_matrix[woman*n+man]],threadIdx.x,label_matrix[woman*n+man],p_star);
					applied_rotations[man*n+label_matrix[woman*n+man]]=true;
				}
			}
		}
		__syncwarp();
	}
	__syncthreads();
	
	if(threadIdx.x==0){
		printf("\nLABEL_MATRIX:\n");
		for(i=0;i<n;i++){
			for(j=0; j<n; j++){
				printf("%i\t",label_matrix[i*n+j]);
			}
			printf("\n");
		}
		printf("\nTriangular matrix:\n");
		for(i=1;i<number_of_rotations;i++){
			for(j=0; j<i; j++){
				printf("%i (%i, %i)\t",triangular_matrix[(i-1)*number_of_rotations+j],i,j);
			}
			printf("\n");
		}
		printf("\nTriangular matrix:\n");
		for (i = 0; i < ((number_of_rotations-1)*number_of_rotations)/2; i++) {
			printf("%i ",triangular_matrix[i]);
		}
		printf("\n");
		free(label_matrix);
		free(is_stable_matrix);
		free(label_second_condition);
		free(applied_rotations);
	}
}
