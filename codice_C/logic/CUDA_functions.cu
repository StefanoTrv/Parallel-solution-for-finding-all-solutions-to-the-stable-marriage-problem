#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "..\data_structures\data_structures.h"

#define false 0
#define true 1


__global__ void build_graph_CUDA(int, int, int*, int*, int*, int*, int*, int*);

__device__ int* label_matrix;
__device__ int* is_stable_matrix;
__device__ int* label_second_condition;
__device__ int* applied_rotations;

__global__ void build_graph_CUDA(int n, int number_of_rotations, int* rotations_vector, int* end_displacement_vector, int* top_matching, int* women_preferences, int* men_preferences, int* triangular_matrix){
	int woman, i, j, first_woman, man, next_woman, k, p_star, iterations;

	if(threadIdx.x==0){
		label_matrix = (int*)malloc(sizeof (int) * n * n);
		is_stable_matrix = (int*)malloc(sizeof (int) * n * n);
		label_second_condition = (int*)malloc(sizeof (int) * n * n);
		applied_rotations = (int*)malloc(sizeof (int) * number_of_rotations);
	}
	__syncthreads();

	for(i=threadIdx.x;i<n;i+=blockDim.x){
		for(j=0; j<n; j++){
			label_matrix[i*n+j]=n;
			is_stable_matrix[j*n+i]=false;
			label_second_condition[j*n+i]=false;
		}
		woman = top_matching[i];
	}
	__syncthreads();

	for(i=threadIdx.x;i<n;i+=blockDim.x){
		is_stable_matrix[top_matching[i]*n+i]=true;
	}
	__syncwarp();

	for (i=threadIdx.x;i<number_of_rotations;i+=blockDim.x){
		applied_rotations[i]=false;
	}
	__syncwarp();

	iterations = number_of_rotations/blockDim.x + ((number_of_rotations%blockDim.x < 1) ? 0 : 1);
	for (i=threadIdx.x;i<iterations;i+=blockDim.x){
		if(i<number_of_rotations){
			if(i==0){
				j=0;
			}else{
				j=end_displacement_vector[i-1]+1;
			}
		}
		__syncwarp();
		if(i<number_of_rotations){
			first_woman=rotations_vector[number_of_rotations+j];
			for(;j<=end_displacement_vector[i];j++){
				man=rotations_vector[j];
				if(j==end_displacement_vector[i]){
					next_woman=first_woman;
				}else{
					next_woman=rotations_vector[number_of_rotations+j+1];
				}
				woman=rotations_vector[number_of_rotations+j];
				//aggiorna rispetto alla donna
				k=n-1;
				while(women_preferences[next_woman*n+k]!=man){
					atomicMin(label_matrix + (next_woman*n+women_preferences[next_woman*n+k]),i);// => label_matrix[next_woman*n+women_preferences[next_woman*n+k]]=i;
					k--;
				}
				//aggiorna rispetto all'uomo
				k=0;
				while(men_preferences[man*n+k]!=next_woman){
					label_second_condition[man*n+men_preferences[man*n+k]]=true;
					k++;
				}

				is_stable_matrix[next_woman*n+man]=true;
				atomicMin(label_matrix + (woman*n+man),i);// => label_matrix[woman*n+man]=i;
			}
		}
		__syncwarp();
	}
	__syncthreads();

	iterations = n/blockDim.x + ((n%blockDim.x < 1) ? 0 : 1);
	for(man=threadIdx.x;man<iterations;man+=blockDim.x){
		if(man<n){
			k=0;
			while(top_matching[man]!=men_preferences[man*n+k]){
				k++;
			}
		}
		__syncwarp();
		if(man<n){
			p_star=-1;
			for(j=k;j<n;j++){
				woman = men_preferences[man*n+j];
				if(label_matrix[woman*n+man]==n) continue;
				if(is_stable_matrix[woman*n+man]){//label di tipo 1
					if(p_star!=-1){
						triangular_matrix[((p_star-1)*p_star)/2+label_matrix[woman*n+man]]=true;
					}
					p_star=label_matrix[woman*n+man];
					applied_rotations[label_matrix[woman*n+man]]=true;
				} else if(!applied_rotations[label_matrix[woman*n+man]] && label_second_condition[man*n+woman]){//label di tipo 2
					triangular_matrix[((label_matrix[woman*n+man]-1)*label_matrix[woman*n+man])/2+p_star]=true;
					applied_rotations[label_matrix[woman*n+man]]=true;
				}
			}
		}
		__syncwarp();
		//resettare applied_rotations
		if(man<n){
			for(j=k;j<n;j++){
				woman=men_preferences[man*n+j];
				applied_rotations[label_matrix[woman*n+man]]=false;
			}
		}
		__syncwarp();
	}
	__syncthreads();
	
	if(threadIdx.x==0){
		free(label_matrix);
		free(is_stable_matrix);
		free(label_second_condition);
		free(applied_rotations);
	}
}
