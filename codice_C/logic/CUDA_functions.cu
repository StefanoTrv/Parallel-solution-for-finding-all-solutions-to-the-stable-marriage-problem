#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "..\data_structures\data_structures.h"

#define false 0
#define true 1


__global__ void build_graph_CUDA(int, int, int*, int*, int*, int*, int*, int*);

__device__ int* label_matrix;
__device__ int* is_stable_matrix;
__device__ int* label_second_condition;
__device__ int* applied_rotations;

__global__ void build_graph_CUDA(int n, int number_of_rotations, int* rotations_vector, int* end_displacement_vector, int* top_matching, int* women_preferences, int* men_preferences, int* triangular_matrix){
	int woman, i, j, first_woman, man, next_woman, k, p_star, iterations;
	if(threadIdx.x==0){
		label_matrix = (int*)malloc(sizeof (int) * n * n);
		is_stable_matrix = (int*)malloc(sizeof (int) * n * n);
		label_second_condition = (int*)malloc(sizeof (int) * n * n);
		applied_rotations = (int*)malloc(sizeof (int) * number_of_rotations * n);
	}
	__syncthreads();

	for(i=threadIdx.x;i<n;i+=blockDim.x){
		for(j=0; j<n; j++){
			label_matrix[i*n+j]=n;
			is_stable_matrix[j*n+i]=false;
			label_second_condition[j*n+i]=false;
		}
		woman = top_matching[i];
	}
	__syncthreads();

	for(i=threadIdx.x;i<n;i+=blockDim.x){
		is_stable_matrix[top_matching[i]*n+i]=true;
	}
	__syncwarp();

	for (i=threadIdx.x;i<number_of_rotations;i+=blockDim.x){
		for(j=0;j<n;j++){
			applied_rotations[j*n+i]=false;
		}
	}
	__syncwarp();

	for (i = threadIdx.x; i < ((n-1)*n)/2; i+=blockDim.x) {
		triangular_matrix[i] = false;
		//printf("%i ",triangular_matrix[i]);
	}
	__syncwarp();

	iterations = number_of_rotations/blockDim.x + ((number_of_rotations%blockDim.x < 1) ? 0 : 1);
	for (i=threadIdx.x;i<iterations*blockDim.x;i+=blockDim.x){
		if(i<number_of_rotations){
			if(i==0){
				j=0;
			}else{
				j=end_displacement_vector[i-1]+1;
			}
			printf("\nThread: %i\ti: %i\tj: %i",threadIdx.x,i,j);
		} else {
			printf("\nThread: %i\ti: %i\tNON ESEGUITO",threadIdx.x,i);
		}
		__syncwarp();
		if(i<number_of_rotations){
			first_woman=rotations_vector[number_of_rotations+j];
			for(;j<=end_displacement_vector[i];j++){
				man=rotations_vector[j];
				if(j==end_displacement_vector[i]){
					next_woman=first_woman;
				}else{
					next_woman=rotations_vector[number_of_rotations+j+1];
				}
				woman=rotations_vector[number_of_rotations+j];
				//aggiorna rispetto alla donna
				k=n-1;
				while(women_preferences[next_woman*n+k]!=man){
					atomicMin(label_matrix + (next_woman*n+women_preferences[next_woman*n+k]),i);// => label_matrix[next_woman*n+women_preferences[next_woman*n+k]]=i;
					printf("\nlabel_matrix[%i] = %i",next_woman*n+women_preferences[next_woman*n+k],label_matrix[next_woman*n+women_preferences[next_woman*n+k]]);
					k--;
				}
				//aggiorna rispetto all'uomo
				k=0;
				while(men_preferences[man*n+k]!=next_woman){
					label_second_condition[man*n+men_preferences[man*n+k]]=true;
					printf("\nlabel_second_condition[%i] = %i",man*n+men_preferences[man*n+k],label_second_condition[man*n+men_preferences[man*n+k]]);
					k++;
				}

				is_stable_matrix[next_woman*n+man]=true;
				atomicMin(label_matrix + (woman*n+man),i);// => label_matrix[woman*n+man]=i;
				printf("\nlabel_matrix[%i] = %i",(woman*n+man),label_matrix[(woman*n+man)]);
			}
		}
		__syncwarp();
	}
	__syncthreads();

	iterations = n/blockDim.x + ((n%blockDim.x < 1) ? 0 : 1);
	for(man=threadIdx.x;man<iterations*blockDim.x;man+=blockDim.x){
		if(man<n){
			k=0;
			while(top_matching[man]!=men_preferences[man*n+k]){
				k++;
			}
			//printf("\n2. Thread: %i\tman: %i\tk: %i\tm_p[]: %i",threadIdx.x,man,k,men_preferences[man*n+k]);
		} else {
			//printf("\n2. Thread: %i\tman: %i\tNON ESEGUITO",threadIdx.x,man);
		}
		__syncwarp();
		if(man<n){
			p_star=-1;
			for(j=k;j<n;j++){
				woman = men_preferences[man*n+j];
				if(label_matrix[woman*n+man]==n) continue;
				if(is_stable_matrix[woman*n+man]){//label di tipo 1
					if(p_star!=-1){
						triangular_matrix[((p_star-1)*p_star)/2+label_matrix[woman*n+man]]=true;
						printf("triangular_matrix[%i] = %i ",((p_star-1)*p_star)/2+label_matrix[woman*n+man],triangular_matrix[((p_star-1)*p_star)/2+label_matrix[woman*n+man]]);
					}
					p_star=label_matrix[woman*n+man];
					applied_rotations[man*n+label_matrix[woman*n+man]]=true;
				} else if(!applied_rotations[man*n+label_matrix[woman*n+man]] && label_second_condition[man*n+woman]){//label di tipo 2
					triangular_matrix[((label_matrix[woman*n+man]-1)*label_matrix[woman*n+man])/2+p_star]=true;
					printf("triangular_matrix[%i] = %i ",((label_matrix[woman*n+man]-1)*label_matrix[woman*n+man])/2+p_star,triangular_matrix[((label_matrix[woman*n+man]-1)*label_matrix[woman*n+man])/2+p_star]);
					applied_rotations[man*n+label_matrix[woman*n+man]]=true;
				}
			}
		}
		__syncwarp();
	}
	__syncthreads();
	
	if(threadIdx.x==0){
		free(label_matrix);
		free(is_stable_matrix);
		free(label_second_condition);
		free(applied_rotations);
	}
}
