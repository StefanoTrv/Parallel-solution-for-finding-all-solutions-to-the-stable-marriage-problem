#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include "..\data_structures\data_structures.h"

#define false 0
#define true 1


__global__ void build_graph_CUDA(int, int, int, int*, int*, int*, int*, int*, int*);

__device__ int* label_matrix;
__device__ char* is_stable_matrix;
__device__ char* label_second_condition;
__device__ int* applied_rotations;
__shared__ int* first_men_preferences_index;
__shared__ int* first_women_preferences_index;

__global__ void build_graph_CUDA(int n, int number_of_rotations, int total_number_of_pairs, int* rotations_vector, int* end_displacement_vector, int* top_matching, int* women_preferences, int* men_preferences, int* triangular_matrix){
	int woman, i, j, first_woman, man, next_woman, k, p_star, iterations;
	if(threadIdx.x==0){
		label_matrix = (int*)malloc(sizeof (int) * n * n);
		is_stable_matrix = (char*)malloc(sizeof (char) * n * n);
		label_second_condition = (char*)malloc(sizeof (char) * n * n);
		applied_rotations = (int*)malloc(sizeof (int) * number_of_rotations * n);
		first_men_preferences_index = (int*)malloc(sizeof (int) * n);
		first_women_preferences_index = (int*)malloc(sizeof (int) * n);
	}
	__syncthreads();

	for(i=threadIdx.x;i<n;i+=blockDim.x){
		for(j=0; j<n; j++){
			label_matrix[i*n+j]=n;
			is_stable_matrix[j*n+i]=false;
			label_second_condition[j*n+i]=false;
		}
		woman = top_matching[i];
	}
	__syncthreads();

	for(i=threadIdx.x;i<n;i+=blockDim.x){
		is_stable_matrix[top_matching[i]*n+i]=true;
		for(j=0;j<n;j++){
			if(top_matching[j]==men_preferences[j*n+i]) first_men_preferences_index[j]=i;
			if(j==women_preferences[top_matching[j]*n+i]) first_women_preferences_index[top_matching[j]]=i;
		}
	}
	__syncwarp();

	for (i=threadIdx.x;i<number_of_rotations;i+=blockDim.x){
		for(j=0;j<n;j++){
			applied_rotations[j*n+i]=false;
		}
	}
	__syncwarp();

	for (i = threadIdx.x; i < ((number_of_rotations-1)*number_of_rotations)/2; i+=blockDim.x) {
		triangular_matrix[i] = false;
	}
	__syncthreads(); //necessario per il terzultimo for

	iterations = number_of_rotations/blockDim.x + ((number_of_rotations%blockDim.x < 1) ? 0 : 1);
	for (i=threadIdx.x;i<iterations*blockDim.x;i+=blockDim.x){
		if(i<number_of_rotations){
			if(i==0){
				j=0;
			}else{
				j=end_displacement_vector[i-1]+1;
			}
		}
		__syncwarp();
		if(i<number_of_rotations){
			first_woman=rotations_vector[total_number_of_pairs+j];
			for(;j<=end_displacement_vector[i];j++){
				man=rotations_vector[j];
				if(j==end_displacement_vector[i]){
					next_woman=first_woman;
				}else{
					next_woman=rotations_vector[total_number_of_pairs+j+1];
				}
				woman=rotations_vector[total_number_of_pairs+j];
				//aggiorna rispetto alla donna
				k=first_women_preferences_index[next_woman]-1;
				//printf("\nkw[%i] = %i",next_woman,k);
				while(women_preferences[next_woman*n+k]!=man){
					atomicMin(label_matrix + (next_woman*n+women_preferences[next_woman*n+k]),i);// => label_matrix[next_woman*n+women_preferences[next_woman*n+k]]=i;
					k--;
				}
				//aggiorna rispetto all'uomo
				k=first_men_preferences_index[man]+1;
				while(men_preferences[man*n+k]!=next_woman){
					label_second_condition[man*n+men_preferences[man*n+k]]=true;
					k++;
				}

				is_stable_matrix[next_woman*n+man]=true;
				atomicMin(label_matrix + (woman*n+man),i);// => label_matrix[woman*n+man]=i;
			}
		}
		__syncwarp();
	}
	__syncthreads();

	iterations = n/blockDim.x + ((n%blockDim.x < 1) ? 0 : 1);
	for(man=threadIdx.x;man<iterations*blockDim.x;man+=blockDim.x){
		if(man<n){
			k=0;
			while(top_matching[man]!=men_preferences[man*n+k]){
				k++;
			}
		}
		__syncwarp();
		if(man<n){
			p_star=-1;
			for(j=k;j<n;j++){
				woman = men_preferences[man*n+j];
				if(label_matrix[woman*n+man]==n) continue;
				if(is_stable_matrix[woman*n+man]){//label di tipo 1
					if(p_star!=-1){
						triangular_matrix[(label_matrix[woman*n+man]-1)*number_of_rotations+p_star]=true;
					}
					p_star=label_matrix[woman*n+man];
					applied_rotations[man*n+label_matrix[woman*n+man]]=true;
				} else if(!applied_rotations[man*n+label_matrix[woman*n+man]] && label_second_condition[man*n+woman]){//label di tipo 2
					triangular_matrix[(p_star-1)*number_of_rotations+label_matrix[woman*n+man]]=true;
					applied_rotations[man*n+label_matrix[woman*n+man]]=true;
				}
			}
		}
		__syncwarp();
	}
	__syncthreads();
	
	if(threadIdx.x==0){
		free(label_matrix);
		free(is_stable_matrix);
		free(label_second_condition);
		free(applied_rotations);
		free(first_men_preferences_index);
		free(first_women_preferences_index);
	}
}
