#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <chrono>
#include <stdint.h>
#include "..\utilities\utilities.h"
#include "..\data_structures\data_structures.h"


#define min(i, j) (((i) < (j)) ? (i) : (j))
#define max(i, j) (((i) > (j)) ? (i) : (j))


static void HandleError( hipError_t err, const char *file, int line ) {
	if (err != hipSuccess) {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
		exit( EXIT_FAILURE );
	}
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


struct ResultsList* all_stable_matchings_CUDA(int n, int* men_preferences, int* women_preferences){
	struct ResultsList* results_list = (struct ResultsList*) malloc(sizeof (struct ResultsList));
	int* top_matching = gale_shapley(n,men_preferences,women_preferences);
	int* inverted_bottom_matching = gale_shapley(n, women_preferences, men_preferences);
	int* bottom_matching = (int*)malloc(sizeof (int) * n);
	for(int i = 0; i < n; i++){
		bottom_matching[inverted_bottom_matching[i]] = i;
	}
	free(inverted_bottom_matching);

	//termina subito se non ci sono rotazioni
	int only_one_matching=1;
	for(int i=0;i<n;i++){
		if(top_matching[i]!=bottom_matching[i]){
			only_one_matching=0;
			break;
		}
	}
	if(only_one_matching){
		results_list->first = (struct ResultsListElement*) malloc(sizeof (struct ResultsListElement));
		results_list->first->value = top_matching;
		results_list->first->next = NULL;
		results_list->last = results_list->first;
		free(bottom_matching);
		return results_list;
	}
	
	//copia top_matching
	int* top_matching_copy = (int*) malloc(sizeof (int) * n);
	for(int i = 0; i < n; i++){
		top_matching_copy[i] = top_matching[i];
	}

	//crea la lista delle rotazioni
	struct RotationsList* rotations_list = find_all_rotations(men_preferences, women_preferences, n, top_matching_copy, bottom_matching);
	free(bottom_matching);
	
	//crea il grafo delle rotazioni

	printf("INIZIO");

	//SEZIONE PARALLELIZZATA
	//creazione delle strutture dati di input
	int number_of_rotations = 0;
	int total_number_of_pairs = 0;
	struct RotationsListElement* list_el = rotations_list->first;
	struct RotationList* rotation_el;
	while(list_el!=NULL){
		number_of_rotations++;
		rotation_el=list_el->value->rotation;
		while(rotation_el!=NULL){
			total_number_of_pairs++;
			rotation_el=rotation_el->next;
		}
		list_el=list_el->next;
	}

	int* rotations_vector;
	HANDLE_ERROR(hipHostAlloc((void**)&rotations_vector, sizeof (int) * total_number_of_pairs * 2, hipHostMallocMapped));
	int* end_displacement_vector;
	HANDLE_ERROR(hipHostAlloc((void**)&end_displacement_vector, sizeof (int) * number_of_rotations, hipHostMallocMapped));
	struct RotationNode** rotation_vector = (struct RotationNode**)malloc(sizeof (struct RotationNode*) * number_of_rotations); //per velocizzare il salvataggio dei risultati

	list_el = rotations_list->first;
	int c1,c2;
	c2 = 0;
	while(list_el!=NULL){
		c1 = 0;
		rotation_vector[list_el->value->index]=list_el->value;//riempio rotation_vector
		rotation_el=list_el->value->rotation;
		while(rotation_el!=NULL){//salva tutte le coppie
			rotations_vector[c1]=rotation_el->man;
			rotations_vector[total_number_of_pairs+c1]=rotation_el->woman;
			c1++;
			rotation_el=rotation_el->next;
		}
		c2+=c1-1;
		end_displacement_vector[list_el->value->index]=c2; //il displacement di questa rotazione
		list_el=list_el->next;
	}
	//preparazione per il lancio del kernel
	int* triangular_matrix, *dev_triangular_matrix, *dev_rotations_vector, *dev_end_displacement_vector, *dev_top_matching, *dev_men_preferences, *dev_women_preferences; 

	HANDLE_ERROR(hipHostAlloc((void**)&triangular_matrix, sizeof (int) * ((n-1)*n)/2, hipHostMallocMapped));
	
	HANDLE_ERROR(hipHostGetDevicePointer(&dev_triangular_matrix, triangular_matrix, 0));
	HANDLE_ERROR(hipHostGetDevicePointer(&dev_rotations_vector, rotations_vector, 0));
	HANDLE_ERROR(hipHostGetDevicePointer(&dev_end_displacement_vector, end_displacement_vector, 0));

	HANDLE_ERROR(hipMalloc((void**)&dev_top_matching, sizeof(int) * n));
	HANDLE_ERROR(hipMalloc((void**)&dev_men_preferences, sizeof(int) * n * n));
	HANDLE_ERROR(hipMalloc((void**)&dev_women_preferences, sizeof(int) * n * n));

	HANDLE_ERROR(hipMemcpy(dev_top_matching, top_matching, sizeof(int) * n, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_men_preferences, men_preferences, sizeof(int) * n * n, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_women_preferences, women_preferences, sizeof(int) * n * n, hipMemcpyHostToDevice));

	printf("\nprima del lancio del kernel\n");
	//lancio del kernel
	int NumThPerBlock = min(max(number_of_rotations, n), 1024);
	build_graph_CUDA<<<1, NumThPerBlock>>>(n, number_of_rotations, dev_rotations_vector, dev_end_displacement_vector,  dev_top_matching, dev_women_preferences, dev_men_preferences, dev_triangular_matrix);
	printf("\ndopo del lancio del kernel\n");

	//libero memoria
	HANDLE_ERROR(hipFree(dev_top_matching));
	printf("\nprima liberazione\n");
	HANDLE_ERROR(hipFree(dev_men_preferences));
	printf("\nseconda liberazione\n");
	HANDLE_ERROR(hipFree(dev_women_preferences));
	printf("\nterza liberazione\n");

	printf("\ndopo la liberazione della memoria\n");

	//applico i risultati alle strutture dati dell'host
	list_el = rotations_list->first;
	int y;
	struct SuccessorsList* sl_el;
	while(list_el!=NULL){
		y=list_el->value->index;
		for(int x = 0; x<y; x++){
			//printf("x: %i   y: %i\n", x, y);
			if(triangular_matrix[((y-1)*y)/2+x]){//se y dipende da x
				list_el->value->missing_predecessors++;//incremento il numero di predecessori di y
				//e aggiungo y tra i successori di x
				sl_el=(struct SuccessorsList*)malloc(sizeof (struct SuccessorsList));
				sl_el->value=list_el->value;
				sl_el=rotation_vector[x]->successors;
				rotation_vector[x]->successors=sl_el;
			}
		}
		list_el=list_el->next;
	}

	//libero memoria
	HANDLE_ERROR(hipHostFree(triangular_matrix));
	HANDLE_ERROR(hipHostFree(rotations_vector));
	HANDLE_ERROR(hipHostFree(end_displacement_vector));
	free(rotation_vector);

	//FINE SEZIONE PARALLELIZZATA
	printf("\nFINE");

	printf("\nCalcolo lista rotazioni libere");
	//calcolo la lista delle rotazioni libere
	struct RotationsList* free_rotations_list = (struct RotationsList*)malloc(sizeof (struct RotationsList));
	free_rotations_list->first=NULL;
	free_rotations_list->last=NULL;
	list_el = rotations_list->first;
	while(list_el!=NULL){
		if(list_el->value->missing_predecessors==0){
			appendRotationsList(free_rotations_list,list_el->value);
		}
		list_el=list_el->next;
	}
	
	printf("\nAggiungo top matching ai risultati");
	//aggiungo top matching ai risultati
	results_list->first = (struct ResultsListElement*) malloc(sizeof (struct ResultsListElement));
	for(int i = 0; i < n; i++){ //per non lavorare sul matching salvato tra i risultati
		top_matching_copy[i] = top_matching[i];
	}
	results_list->first->value = top_matching_copy;
	results_list->first->next = NULL;
	results_list->last = results_list->first;
	
	printf("\nRicerca ricorsiva dei risultati");
	if(rotations_list->first != NULL){
		recursive_search(top_matching, n, free_rotations_list->first, results_list);
	}
		
	printf("\nUltimi free");
	free(top_matching);
	free_rotations_list_struct(rotations_list);
	list_el=free_rotations_list->first;
	struct RotationsListElement* temp;
	while(list_el!=NULL){
		temp=list_el;
		list_el=list_el->next;
		free(temp);
	}
	free(free_rotations_list);
	return results_list;
}



/*struct ResultsList* all_stable_matchings_times_CUDA(int n, int* men_preferences, int* women_preferences, int* time_gale_shapley, int* time_find_all_rotations, int* time_build_graph, int* time_recursive){
	// Time measure
    std::chrono::steady_clock::time_point start_time;
    std::chrono::steady_clock::time_point end_time;

	struct ResultsList* results_list = (struct ResultsList*) malloc(sizeof (struct ResultsList));
	
	start_time = std::chrono::steady_clock::now();
	int* top_matching = gale_shapley(n,men_preferences,women_preferences);
	end_time = std::chrono::steady_clock::now();
	*time_gale_shapley = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();

	int* inverted_bottom_matching = gale_shapley(n, women_preferences, men_preferences);
	int* bottom_matching = (int*)malloc(sizeof (int) * n);
	for(int i = 0; i < n; i++){
		bottom_matching[inverted_bottom_matching[i]] = i;
	}
	free(inverted_bottom_matching);

	//termina subito se non ci sono rotazioni
	int only_one_matching=1;
	for(int i=0;i<n;i++){
		if(top_matching[i]!=bottom_matching[i]){
			only_one_matching=0;
			break;
		}
	}
	if(only_one_matching){
		results_list->first = (struct ResultsListElement*) malloc(sizeof (struct ResultsListElement));
		results_list->first->value = top_matching;
		results_list->first->next = NULL;
		results_list->last = results_list->first;
		free(bottom_matching);
		return results_list;
	}
	
	//copia top_matching
	int* top_matching_copy = (int*) malloc(sizeof (int) * n);
	for(int i = 0; i < n; i++){
		top_matching_copy[i] = top_matching[i];
	}

	//crea la lista delle rotazioni
	start_time = std::chrono::steady_clock::now();
	struct RotationsList* rotations_list = find_all_rotations(men_preferences, women_preferences, n, top_matching_copy,bottom_matching);
	free(bottom_matching);
	end_time = std::chrono::steady_clock::now();
	*time_find_all_rotations = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();

	
	//crea il grafo delle rotazioni
	start_time = std::chrono::steady_clock::now();
	build_graph(n, rotations_list, top_matching, men_preferences, women_preferences);
	end_time = std::chrono::steady_clock::now();
	*time_build_graph = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();

	//calcolo la lista delle rotazioni libere
	struct RotationsList* free_rotations_list = (struct RotationsList*)malloc(sizeof (struct RotationsList));
	free_rotations_list->first=NULL;
	free_rotations_list->last=NULL;
	struct RotationsListElement* list_el = rotations_list->first;
	while(list_el!=NULL){
		if(list_el->value->missing_predecessors==0){
			appendRotationsList(free_rotations_list,list_el->value);
		}
		list_el=list_el->next;
	}
	
	//aggiungo top matching ai risultati
	results_list->first = (struct ResultsListElement*) malloc(sizeof (struct ResultsListElement));
	for(int i = 0; i < n; i++){ //per non lavorare sul matching salvato tra i risultati
		top_matching_copy[i] = top_matching[i];
	}
	results_list->first->value = top_matching_copy;
	results_list->first->next = NULL;
	results_list->last = results_list->first;

	if(rotations_list->first != NULL){
		start_time = std::chrono::steady_clock::now();
		recursive_search(top_matching, n, free_rotations_list->first, results_list);
		end_time = std::chrono::steady_clock::now();
		*time_recursive = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();
	}
	
	free(top_matching);
	free_rotations_list_struct(rotations_list);
	list_el=free_rotations_list->first;
	struct RotationsListElement* temp;
	while(list_el!=NULL){
		temp=list_el;
		list_el=list_el->next;
		free(temp);
	}
	free(free_rotations_list);
	return results_list;
}*/