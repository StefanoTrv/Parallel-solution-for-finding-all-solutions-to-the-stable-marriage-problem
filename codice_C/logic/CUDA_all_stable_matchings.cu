#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <chrono>
#include <stdint.h>
#include "..\utilities\utilities.h"
#include "..\data_structures\data_structures.h"


#define min(i, j) (((i) < (j)) ? (i) : (j))
#define max(i, j) (((i) > (j)) ? (i) : (j))


struct ResultsList* all_stable_matchings_CUDA(int n, int* men_preferences, int* women_preferences){
	struct ResultsList* results_list = (struct ResultsList*) malloc(sizeof (struct ResultsList));
	int* top_matching = gale_shapley(n,men_preferences,women_preferences);
	int* inverted_bottom_matching = gale_shapley(n, women_preferences, men_preferences);
	int* bottom_matching = (int*)malloc(sizeof (int) * n);
	for(int i = 0; i < n; i++){
		bottom_matching[inverted_bottom_matching[i]] = i;
	}
	free(inverted_bottom_matching);

	//termina subito se non ci sono rotazioni
	int only_one_matching=1;
	for(int i=0;i<n;i++){
		if(top_matching[i]!=bottom_matching[i]){
			only_one_matching=0;
			break;
		}
	}
	if(only_one_matching){
		results_list->first = (struct ResultsListElement*) malloc(sizeof (struct ResultsListElement));
		results_list->first->value = top_matching;
		results_list->first->next = NULL;
		results_list->last = results_list->first;
		free(bottom_matching);
		return results_list;
	}
	
	//copia top_matching
	int* top_matching_copy = (int*) malloc(sizeof (int) * n);
	for(int i = 0; i < n; i++){
		top_matching_copy[i] = top_matching[i];
	}

	//crea la lista delle rotazioni
	struct RotationsList* rotations_list = find_all_rotations(men_preferences, women_preferences, n, top_matching_copy, bottom_matching);
	free(bottom_matching);
	
	//crea il grafo delle rotazioni

	//SEZIONE PARALLELIZZATA
	//creazione delle strutture dati di input
	int number_of_rotations = 0;
	int total_number_of_pairs = 0;
	struct RotationsListElement* list_el = rotations_list->first;
	struct RotationList* rotation_el;
	while(list_el!=NULL){
		number_of_rotations++;
		rotation_el=list_el->value->rotation;
		while(rotation_el!=NULL){
			total_number_of_pairs++;
			rotation_el=rotation_el->next;
		}
		list_el=list_el->next;
	}

	int* rotations_vector = (int*)malloc(sizeof (int) * total_number_of_pairs * 2);
	int* end_displacement_vector = (int*)malloc(sizeof (int) * number_of_rotations);
	struct RotationNode** rotation_vector = (struct RotationNode**)malloc(sizeof (struct RotationNode*) * number_of_rotations); //per velocizzare il salvataggio dei risultati

	list_el = rotations_list->first;
	int c1,c2;
	while(list_el!=NULL){
		c1 = 0;
		rotation_vector[list_el->value->index]=list_el->value;//riempio rotation_vector
		rotation_el=list_el->value->rotation;
		while(rotation_el!=NULL){//salva tutte le coppie
			rotations_vector[c1]=rotation_el->man;
			rotations_vector[total_number_of_pairs+c1]=rotation_el->woman;
			c1++;
			rotation_el=rotation_el->next;
		}
		c2+=c1-1;
		end_displacement_vector[list_el->value->index]=c2; //il displacement di questa rotazione
		list_el=list_el->next;
	}

	//Alloco memoria page-locked sull'host
	int* triangular_matrix;
	hipHostAlloc((void**)&triangular_matrix, sizeof (int) * ((n-1)*n)/2, hipHostMallocDefault); //da vedere se passare a mapped
	
	//Sposto i dati su device
	void* dev_ptr = triangular_matrix; 
	hipMemcpy(dev_ptr, triangular_matrix, sizeof (int) * ((n-1)*n)/2, hipMemcpyHostToDevice);

	//lancio del kernel
	int NumThPerBlock = min(max(number_of_rotations, n), 1024);
	build_graph_CUDA<<<1, NumThPerBlock>>>(n, number_of_rotations, rotations_vector, end_displacement_vector,  top_matching, women_preferences, men_preferences, dev_ptr);
	
	//Sposto i risultati sull'host
	hipMemcpy(triangular_matrix, dev_ptr, sizeof (int) * ((n-1)*n)/2, hipMemcpyDeviceToHost);

	//libero la memoria sul device
	hipFree(dev_ptr);

	//applico i risultati alle strutture dati dell'host
	list_el = rotations_list->first;
	int y;
	struct SuccessorsList* sl_el;
	while(list_el!=NULL){
		y=list_el->value->index;
		for(int x = 0; x<y; x++){
			if(triangular_matrix[((y-1)*y)/2+x]){//se y dipende da x
				list_el->value->missing_predecessors++;//incremento il numero di predecessori di y
				//e aggiungo y tra i successori di x
				sl_el=(struct SuccessorsList*)malloc(sizeof (struct SuccessorsList));
				sl_el->value=list_el->value;
				sl_el=rotation_vector[x]->successors;
				rotation_vector[x]->successors=sl_el;
			}
		}
		list_el=list_el->next;
	}

	//libero memoria
	hipHostFree(triangular_matrix);
	free(rotations_vector);
	free(end_displacement_vector);
	free(rotation_vector);

	//FINE SEZIONE PARALLELIZZATA

	
	//calcolo la lista delle rotazioni libere
	struct RotationsList* free_rotations_list = (struct RotationsList*)malloc(sizeof (struct RotationsList));
	free_rotations_list->first=NULL;
	free_rotations_list->last=NULL;
	list_el = rotations_list->first;
	while(list_el!=NULL){
		if(list_el->value->missing_predecessors==0){
			appendRotationsList(free_rotations_list,list_el->value);
		}
		list_el=list_el->next;
	}
	
	//aggiungo top matching ai risultati
	results_list->first = (struct ResultsListElement*) malloc(sizeof (struct ResultsListElement));
	for(int i = 0; i < n; i++){ //per non lavorare sul matching salvato tra i risultati
		top_matching_copy[i] = top_matching[i];
	}
	results_list->first->value = top_matching_copy;
	results_list->first->next = NULL;
	results_list->last = results_list->first;

	if(rotations_list->first != NULL){
		recursive_search(top_matching, n, free_rotations_list->first, results_list);
	}
	
	free(top_matching);
	free_rotations_list_struct(rotations_list);
	list_el=free_rotations_list->first;
	struct RotationsListElement* temp;
	while(list_el!=NULL){
		temp=list_el;
		list_el=list_el->next;
		free(temp);
	}
	free(free_rotations_list);
	return results_list;
}



/*struct ResultsList* all_stable_matchings_times_CUDA(int n, int* men_preferences, int* women_preferences, int* time_gale_shapley, int* time_find_all_rotations, int* time_build_graph, int* time_recursive){
	// Time measure
    std::chrono::steady_clock::time_point start_time;
    std::chrono::steady_clock::time_point end_time;

	struct ResultsList* results_list = (struct ResultsList*) malloc(sizeof (struct ResultsList));
	
	start_time = std::chrono::steady_clock::now();
	int* top_matching = gale_shapley(n,men_preferences,women_preferences);
	end_time = std::chrono::steady_clock::now();
	*time_gale_shapley = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();

	int* inverted_bottom_matching = gale_shapley(n, women_preferences, men_preferences);
	int* bottom_matching = (int*)malloc(sizeof (int) * n);
	for(int i = 0; i < n; i++){
		bottom_matching[inverted_bottom_matching[i]] = i;
	}
	free(inverted_bottom_matching);

	//termina subito se non ci sono rotazioni
	int only_one_matching=1;
	for(int i=0;i<n;i++){
		if(top_matching[i]!=bottom_matching[i]){
			only_one_matching=0;
			break;
		}
	}
	if(only_one_matching){
		results_list->first = (struct ResultsListElement*) malloc(sizeof (struct ResultsListElement));
		results_list->first->value = top_matching;
		results_list->first->next = NULL;
		results_list->last = results_list->first;
		free(bottom_matching);
		return results_list;
	}
	
	//copia top_matching
	int* top_matching_copy = (int*) malloc(sizeof (int) * n);
	for(int i = 0; i < n; i++){
		top_matching_copy[i] = top_matching[i];
	}

	//crea la lista delle rotazioni
	start_time = std::chrono::steady_clock::now();
	struct RotationsList* rotations_list = find_all_rotations(men_preferences, women_preferences, n, top_matching_copy,bottom_matching);
	free(bottom_matching);
	end_time = std::chrono::steady_clock::now();
	*time_find_all_rotations = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();

	
	//crea il grafo delle rotazioni
	start_time = std::chrono::steady_clock::now();
	build_graph(n, rotations_list, top_matching, men_preferences, women_preferences);
	end_time = std::chrono::steady_clock::now();
	*time_build_graph = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();

	//calcolo la lista delle rotazioni libere
	struct RotationsList* free_rotations_list = (struct RotationsList*)malloc(sizeof (struct RotationsList));
	free_rotations_list->first=NULL;
	free_rotations_list->last=NULL;
	struct RotationsListElement* list_el = rotations_list->first;
	while(list_el!=NULL){
		if(list_el->value->missing_predecessors==0){
			appendRotationsList(free_rotations_list,list_el->value);
		}
		list_el=list_el->next;
	}
	
	//aggiungo top matching ai risultati
	results_list->first = (struct ResultsListElement*) malloc(sizeof (struct ResultsListElement));
	for(int i = 0; i < n; i++){ //per non lavorare sul matching salvato tra i risultati
		top_matching_copy[i] = top_matching[i];
	}
	results_list->first->value = top_matching_copy;
	results_list->first->next = NULL;
	results_list->last = results_list->first;

	if(rotations_list->first != NULL){
		start_time = std::chrono::steady_clock::now();
		recursive_search(top_matching, n, free_rotations_list->first, results_list);
		end_time = std::chrono::steady_clock::now();
		*time_recursive = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();
	}
	
	free(top_matching);
	free_rotations_list_struct(rotations_list);
	list_el=free_rotations_list->first;
	struct RotationsListElement* temp;
	while(list_el!=NULL){
		temp=list_el;
		list_el=list_el->next;
		free(temp);
	}
	free(free_rotations_list);
	return results_list;
}*/