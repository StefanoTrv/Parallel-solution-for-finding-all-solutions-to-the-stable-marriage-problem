#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "..\utilities\utilities.h"
#include "..\data_structures\data_structures.h"
#include "serial_functions.h"


#define min(i, j) (((i) < (j)) ? (i) : (j))
#define max(i, j) (((i) > (j)) ? (i) : (j))


static void HandleError( hipError_t err, const char *file, int line ) {
	if (err != hipSuccess) {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
		exit( EXIT_FAILURE );
	}
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


struct ResultsList* all_stable_matchings_CUDA(int n, int* men_preferences, int* women_preferences){
	struct ResultsList* results_list = (struct ResultsList*) malloc(sizeof (struct ResultsList));
	int* top_matching = gale_shapley(n,men_preferences,women_preferences);
	int* inverted_bottom_matching = gale_shapley(n, women_preferences, men_preferences);
	int* bottom_matching = (int*)malloc(sizeof (int) * n);
	for(int i = 0; i < n; i++){
		bottom_matching[inverted_bottom_matching[i]] = i;
	}
	free(inverted_bottom_matching);

	//termina subito se non ci sono rotazioni
	int only_one_matching=1;
	for(int i=0;i<n;i++){
		if(top_matching[i]!=bottom_matching[i]){
			only_one_matching=0;
			break;
		}
	}
	if(only_one_matching){
		results_list->first = (struct ResultsListElement*) malloc(sizeof (struct ResultsListElement));
		results_list->first->value = top_matching;
		results_list->first->next = NULL;
		results_list->last = results_list->first;
		free(bottom_matching);
		return results_list;
	}
	
	//copia top_matching
	int* top_matching_copy = (int*) malloc(sizeof (int) * n);
	for(int i = 0; i < n; i++){
		top_matching_copy[i] = top_matching[i];
	}

	//crea la lista delle rotazioni
	struct RotationsList* rotations_list = find_all_rotations(men_preferences, women_preferences, n, top_matching_copy, bottom_matching);
	free(bottom_matching);
	
	//crea il grafo delle rotazioni

	//printf("INIZIO");

	//SEZIONE PARALLELIZZATA
	//creazione delle strutture dati di input
	int number_of_rotations = 0;
	int total_number_of_pairs = 0;
	struct RotationsListElement* list_el = rotations_list->first;
	struct RotationList* rotation_el;
	while(list_el!=NULL){
		number_of_rotations++;
		rotation_el=list_el->value->rotation;
		while(rotation_el!=NULL){
			total_number_of_pairs++;
			rotation_el=rotation_el->next;
		}
		list_el=list_el->next;
	}

	int* rotations_vector;
	HANDLE_ERROR(hipHostAlloc((void**)&rotations_vector, sizeof (int) * total_number_of_pairs * 2, hipHostMallocMapped));
	int* end_displacement_vector;
	HANDLE_ERROR(hipHostAlloc((void**)&end_displacement_vector, sizeof (int) * number_of_rotations, hipHostMallocMapped));
	struct RotationNode** rotation_vector = (struct RotationNode**)malloc(sizeof (struct RotationNode*) * number_of_rotations); //per velocizzare il salvataggio dei risultati

	list_el = rotations_list->first;
	int c1 = 0;
	int c2 = 0;
	while(list_el!=NULL){
		rotation_vector[list_el->value->index]=list_el->value;//riempio rotation_vector
		rotation_el=list_el->value->rotation;
		while(rotation_el!=NULL){//salva tutte le coppie
			rotations_vector[c1]=rotation_el->man;
			rotations_vector[total_number_of_pairs+c1]=rotation_el->woman;
			c1++;
			rotation_el=rotation_el->next;
		}
		c2=c1-1;
		end_displacement_vector[list_el->value->index]=c2; //il displacement di questa rotazione
		list_el=list_el->next;
	}

	/*printf("\nVettore delle rotazioni:\n");
	for(int i=0;i<total_number_of_pairs;i++){
		printf("(%i,%i)",rotations_vector[i],rotations_vector[total_number_of_pairs+i]);
	}
	printf("\n");
	printf("\nVettore dei displacement:\n");
	for(int i=0;i<number_of_rotations;i++){
		printf("%i ",end_displacement_vector[i]);
	}
	printf("\n");*/

	//preparazione per il lancio del kernel
	int* triangular_matrix, *dev_triangular_matrix, *dev_rotations_vector, *dev_end_displacement_vector, *dev_top_matching, *dev_men_preferences, *dev_women_preferences;

	HANDLE_ERROR(hipHostAlloc((void**)&triangular_matrix, sizeof (int) * ((number_of_rotations-1)*number_of_rotations)/2, hipHostMallocMapped));
	
	if(number_of_rotations>1){
		(hipHostGetDevicePointer(&dev_triangular_matrix, triangular_matrix, 0));
	} else{
		dev_triangular_matrix=NULL;
	}
	HANDLE_ERROR(hipHostGetDevicePointer(&dev_rotations_vector, rotations_vector, 0));
	HANDLE_ERROR(hipHostGetDevicePointer(&dev_end_displacement_vector, end_displacement_vector, 0));

	HANDLE_ERROR(hipMalloc((void**)&dev_top_matching, sizeof(int) * n));
	HANDLE_ERROR(hipMalloc((void**)&dev_men_preferences, sizeof(int) * n * n));
	HANDLE_ERROR(hipMalloc((void**)&dev_women_preferences, sizeof(int) * n * n));

	HANDLE_ERROR(hipMemcpy(dev_top_matching, top_matching, sizeof(int) * n, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_men_preferences, men_preferences, sizeof(int) * n * n, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_women_preferences, women_preferences, sizeof(int) * n * n, hipMemcpyHostToDevice));

	//printf("\nprima del lancio del kernel\n");
	//lancio del kernel
	int NumThPerBlock = min(max(number_of_rotations, n), 1024);
	size_t free_t,total_t;
	hipMemGetInfo(&free_t,&total_t);
	hipDeviceSetLimit(hipLimitMallocHeapSize, free_t * 0.9);
	build_graph_CUDA<<<1, NumThPerBlock>>>(n, number_of_rotations, total_number_of_pairs, dev_rotations_vector, dev_end_displacement_vector,  dev_top_matching, dev_women_preferences, dev_men_preferences, dev_triangular_matrix);
	//printf("\ndopo del lancio del kernel\n");

	//libero memoria
	hipDeviceSynchronize();
	HANDLE_ERROR(hipFree(dev_top_matching));
	//printf("\nprima liberazione\n");
	HANDLE_ERROR(hipFree(dev_men_preferences));
	//printf("\nseconda liberazione\n");
	HANDLE_ERROR(hipFree(dev_women_preferences));
	//printf("\nterza liberazione\n");

	//printf("\ndopo la liberazione della memoria\n");

	//applico i risultati alle strutture dati dell'host
	list_el = rotations_list->first;
	int y;
	struct SuccessorsList* sl_el;
	while(list_el!=NULL){
		//printf("\\_/");
		y=list_el->value->index;
		for(int x = 0; x<y; x++){
			//printf("x: %i\ty: %i\ttriangular_matrix[%i] = %i\n", x, y,(y-1)*number_of_rotations+x,triangular_matrix[(y-1)*number_of_rotations+x]);
			if(triangular_matrix[(y-1)*number_of_rotations+x]){//se y dipende da x
				//printf("aggiungo dipendenza...");
				list_el->value->missing_predecessors++;//incremento il numero di predecessori di y
				//printf("\t%i\t",list_el->value->missing_predecessors);
				//e aggiungo y tra i successori di x
				sl_el=(struct SuccessorsList*)malloc(sizeof (struct SuccessorsList));
				sl_el->value=list_el->value;
				//printf("\t%i\t",sl_el->value->index);
				sl_el->next=rotation_vector[x]->successors;
				//printf("\t%i-%i\t",sl_el->next->value->index,rotation_vector[x]->successors->value->index);
				rotation_vector[x]->successors=sl_el;
				//printf("\t%i-%i\t",rotation_vector[x]->successors->value->index,sl_el->value->index);
				//printf("\n");
			}
		}
		list_el=list_el->next;
	}

	//libero memoria
	HANDLE_ERROR(hipHostFree(triangular_matrix));
	HANDLE_ERROR(hipHostFree(rotations_vector));
	HANDLE_ERROR(hipHostFree(end_displacement_vector));
	free(rotation_vector);

	//FINE SEZIONE PARALLELIZZATA
	//printf("\nFINE");

	//printf("\nCalcolo lista rotazioni libere");
	//calcolo la lista delle rotazioni libere
	struct RotationsList* free_rotations_list = (struct RotationsList*)malloc(sizeof (struct RotationsList));
	free_rotations_list->first=NULL;
	free_rotations_list->last=NULL;
	list_el = rotations_list->first;
	while(list_el!=NULL){
		if(list_el->value->missing_predecessors==0){
			appendRotationsList(free_rotations_list,list_el->value);
		}
		list_el=list_el->next;
	}

	//TEST
	/*
	struct RotationsListElement* lel = rotations_list->first;
	struct RotationList* le;
	struct SuccessorsList* sl;
	printf("GRAFO PARALLELO:\n");
	while(lel!=NULL){
		le=lel->value->rotation;
		while(le!=NULL){
			printf("(%i,%i), ",le->man,le->woman);
			le=le->next;
		}
		printf("ha indice %i e ",lel->value->index);
		if(lel->value->missing_predecessors>0) printf("non ");
		printf("e' una rotazione libera");
		if(lel->value->missing_predecessors>0){
			printf(" con %i predecessori: ",lel->value->missing_predecessors);
			sl=lel->value->successors;
			printf(" con %i predecessori e i seguenti successori: ",lel->value->missing_predecessors);
			sl=lel->value->successors;
			while(sl!=NULL) {
				printf("%i, ",sl->value->index);
				sl=sl->next;
			}
		}
		printf("\n");
		lel = lel->next;
	}
	printf("\n\n");*/
	//TEST

	
	//printf("\nAggiungo top matching ai risultati");
	//aggiungo top matching ai risultati
	results_list->first = (struct ResultsListElement*) malloc(sizeof (struct ResultsListElement));
	for(int i = 0; i < n; i++){ //per non lavorare sul matching salvato tra i risultati
		top_matching_copy[i] = top_matching[i];
	}
	results_list->first->value = top_matching_copy;
	results_list->first->next = NULL;
	results_list->last = results_list->first;
	
	//printf("\nRicerca ricorsiva dei risultati");
	if(rotations_list->first != NULL){
		recursive_search(top_matching, n, free_rotations_list->first, results_list);
	}
		
	//printf("\nUltimi free\n");
	free(top_matching);
	free_rotations_list_struct(rotations_list);
	list_el=free_rotations_list->first;
	struct RotationsListElement* temp;
	while(list_el!=NULL){
		temp=list_el;
		list_el=list_el->next;
		free(temp);
	}
	free(free_rotations_list);
	return results_list;
}



/*struct ResultsList* all_stable_matchings_times_CUDA(int n, int* men_preferences, int* women_preferences, int* time_gale_shapley, int* time_find_all_rotations, int* time_build_graph, int* time_recursive){
	// Time measure
    std::chrono::steady_clock::time_point start_time;
    std::chrono::steady_clock::time_point end_time;

	struct ResultsList* results_list = (struct ResultsList*) malloc(sizeof (struct ResultsList));
	
	start_time = std::chrono::steady_clock::now();
	int* top_matching = gale_shapley(n,men_preferences,women_preferences);
	end_time = std::chrono::steady_clock::now();
	*time_gale_shapley = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();

	int* inverted_bottom_matching = gale_shapley(n, women_preferences, men_preferences);
	int* bottom_matching = (int*)malloc(sizeof (int) * n);
	for(int i = 0; i < n; i++){
		bottom_matching[inverted_bottom_matching[i]] = i;
	}
	free(inverted_bottom_matching);

	//termina subito se non ci sono rotazioni
	int only_one_matching=1;
	for(int i=0;i<n;i++){
		if(top_matching[i]!=bottom_matching[i]){
			only_one_matching=0;
			break;
		}
	}
	if(only_one_matching){
		results_list->first = (struct ResultsListElement*) malloc(sizeof (struct ResultsListElement));
		results_list->first->value = top_matching;
		results_list->first->next = NULL;
		results_list->last = results_list->first;
		free(bottom_matching);
		return results_list;
	}
	
	//copia top_matching
	int* top_matching_copy = (int*) malloc(sizeof (int) * n);
	for(int i = 0; i < n; i++){
		top_matching_copy[i] = top_matching[i];
	}

	//crea la lista delle rotazioni
	start_time = std::chrono::steady_clock::now();
	struct RotationsList* rotations_list = find_all_rotations(men_preferences, women_preferences, n, top_matching_copy,bottom_matching);
	free(bottom_matching);
	end_time = std::chrono::steady_clock::now();
	*time_find_all_rotations = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();

	
	//crea il grafo delle rotazioni
	start_time = std::chrono::steady_clock::now();
	build_graph(n, rotations_list, top_matching, men_preferences, women_preferences);
	end_time = std::chrono::steady_clock::now();
	*time_build_graph = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();

	//calcolo la lista delle rotazioni libere
	struct RotationsList* free_rotations_list = (struct RotationsList*)malloc(sizeof (struct RotationsList));
	free_rotations_list->first=NULL;
	free_rotations_list->last=NULL;
	struct RotationsListElement* list_el = rotations_list->first;
	while(list_el!=NULL){
		if(list_el->value->missing_predecessors==0){
			appendRotationsList(free_rotations_list,list_el->value);
		}
		list_el=list_el->next;
	}
	
	//aggiungo top matching ai risultati
	results_list->first = (struct ResultsListElement*) malloc(sizeof (struct ResultsListElement));
	for(int i = 0; i < n; i++){ //per non lavorare sul matching salvato tra i risultati
		top_matching_copy[i] = top_matching[i];
	}
	results_list->first->value = top_matching_copy;
	results_list->first->next = NULL;
	results_list->last = results_list->first;

	if(rotations_list->first != NULL){
		start_time = std::chrono::steady_clock::now();
		recursive_search(top_matching, n, free_rotations_list->first, results_list);
		end_time = std::chrono::steady_clock::now();
		*time_recursive = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();
	}
	
	free(top_matching);
	free_rotations_list_struct(rotations_list);
	list_el=free_rotations_list->first;
	struct RotationsListElement* temp;
	while(list_el!=NULL){
		temp=list_el;
		list_el=list_el->next;
		free(temp);
	}
	free(free_rotations_list);
	return results_list;
}*/